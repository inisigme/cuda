#include "stdafx.h"
#include "cuda_includes.h"
#include "utils.h"
#include "tests.cuh"
#include "time.h"

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);

	gpuErrchk(hipMalloc((void**)&dev_c, size * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_a, size * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&dev_b, size * sizeof(int)));


	gpuErrchk(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice));


	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost));

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	return cudaStatus;
}


int main(int argc, char *argv[])
{
	try {
		const unsigned int size = 1000;
		std::vector<int> a(size);
		std::vector<int> b(size);
		std::vector<int> c(size,0);

		for (auto i : a) i = rand();
		for (auto i : b) i = rand();
		

		gpuErrchk(addWithCuda(&c[0], &a[0], &b[0], size));


		gpuErrchk(hipDeviceReset());

		std::cout << "end" << std::endl;
		getchar();
		return 0;
	}
	catch (std::exception& exc) {
		cerr << exc.what() << std::endl;
		getchar();
		exit(EXIT_FAILURE);
	}
	return (EXIT_SUCCESS);
}