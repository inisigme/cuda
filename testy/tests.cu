#include "hip/hip_runtime.h"
#include "tests.cuh"

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	c[i] = a[i] + b[i];
}